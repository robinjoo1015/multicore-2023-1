
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>

using namespace std;
using namespace std::chrono;

#define N 10

__global__ void add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%10;
	}
}

int main(void) 
{
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);
	
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	auto start = high_resolution_clock::now();

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	// Launch add() kernel on GPU with N blocks
	add<<<N,1>>>(d_a, d_b, d_c);
	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	// Cleanup

	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
 
    	cout << "Time taken by function: "
         << duration.count() << " microseconds" << endl;

	for (int i=0;i<N;i++) {
		printf("a[%d]=%d , b[%d]=%d, c[%d]=%d\n",i,a[i],i,b[i],i,c[i]);
	}

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
